#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <assert.h>
#include "ising.h"

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

__global__ void gpu_get_neighors(int *neighbors, int n , int k)
{
	for (int off1 = 0; off1 < n/gridDim.x+1 ; off1++)
	{
		for(int off2 = 0; off2 < n/blockDim.x+1 ;off2++){
			
			int m = blockIdx.x+off1*gridDim.x;
			int l = threadIdx.x+off2*blockDim.x;

			int counter_i =0;
			if(m<n && l<n){
				for (int i = m-(k/2); i <= m+(k/2); i++)
				{
					int counter_j=0;
					for (int j = l-(k/2); j <= l+(k/2); j++)
					{
						int index , index_i , index_j;
						index = m*n*k*k + l*k*k + counter_i*k +counter_j;
						index_i =(n+i)%n;
						index_j=(n+j)%n;
						neighbors[index] = index_i*n+index_j;
						counter_j++; 
					}
					counter_i++;
				}
			}
		}
	}
}

__global__ void gpu_update_sign(int *G, double *w ,int *neighbors , int k , int n ,int *temp, int *flag,int it_b ,int it_t)
{
	int result;
	double sum = 0.0;
	int buf=0;
	//Find the indexes
	int x = blockIdx.x+it_b*gridDim.x;
	int y = threadIdx.x+it_t*blockDim.x;

	if (blockIdx.x+it_b*gridDim.x<n && threadIdx.x+it_t*blockDim.x<n)
	{
		//Calculate result	
		for (int i = 0; i < k; i++){
			for (int j = 0; j < k; j++){
				sum += ((double)G[neighbors[x*n*k*k+y*k*k+i*k+j]])*w[i*k+j];	
			}
		}
		//Evaluate and write back
		if ( sum > 1e-6){
			result = 1; 
			if (result != G[neighbors[x*n*k*k+y*k*k+12]])
				buf++;
		}
		else if( sum < -(1e-6)){
			result = -1;
			if (result != G[neighbors[x*n*k*k+y*k*k+12]])
				buf++;
		}
		else
			result = G[neighbors[x*n*k*k+y*k*k+12]];
	
		*flag+=buf;
		temp[x*n+y] =result;	
	}
}

void ising_parallel(int *G, double *w, int k, int n)
{	
	//Setting number of of threads and blocks
	int number_of_threads = 1024;
	int number_of_block = 256;
	
	//Calculation the number of the itterarions that they will be needed
	double it_b = (double) n / (double) number_of_block;
	it_b = (ceil(it_b));
	double it_t = (double) n / (double) number_of_threads;
	it_t = (ceil(it_t));
	
	//Initiate the parameters in the memory of the device 
	int *G_cuda , *neighbors_cuda, *temp;
	hipMalloc((void **) &G_cuda,sizeof(int)*n*n);
	hipMalloc((void **) &neighbors_cuda,sizeof(int)*n*n*25);
	hipMalloc((void **) &temp , sizeof(int)*n*n);
	
	double *w_cuda;
	hipMalloc((void **) &w_cuda,sizeof(double)*5*5);
	
	int *flag_cuda , *flag;
	flag = (int *) malloc(sizeof(int));
	hipMalloc((void **) &flag_cuda , sizeof(int));
	
	hipMemcpyAsync(G_cuda,G,sizeof(int)*n*n,hipMemcpyHostToDevice,NULL);
	hipMemcpyAsync(w_cuda,w,sizeof(double)*5*5,hipMemcpyHostToDevice,NULL);
	
	//Calculation the indexes of the neighbors in parallel
	gpu_get_neighors<<<number_of_block,number_of_threads>>>(neighbors_cuda , n , 5);
	checkCuda( hipGetLastError());
	hipDeviceSynchronize();

	//Starting the k times itteration
	int counter=0;
	for (int i = 0; i < k; i++)
	{
		//Calling it_b*it_t times the kernel function to make the complete update
		int buff=0;
		for(int j=0; j<(int)it_b;j++)
		{
			for (int l = 0; l< (int)it_t; l++)
			{
				gpu_update_sign<<<number_of_block,number_of_threads>>>(G_cuda,w_cuda,neighbors_cuda,5,n,temp,flag_cuda,j,l);
				checkCuda( hipGetLastError());
				checkCuda( hipMemcpy(flag,flag_cuda,sizeof(int),hipMemcpyDeviceToHost) );
				buff+=flag[0];
				counter++;
			}
		}
		//Setting data ready for the next itteration
		checkCuda( hipMemcpy(G_cuda,temp,sizeof(int)*n*n,hipMemcpyDeviceToDevice) );	

		//Check if there is no changes
		if(buff==0){
			printf("breakpoint %ds:\n",counter);
			break;
		}
	}

	//Writing the final result
	checkCuda(hipMemcpy(G,G_cuda,sizeof(int)*n*n,hipMemcpyDeviceToHost) );

	hipFree(G_cuda);
	hipFree(w_cuda);
	hipFree(neighbors_cuda);
	hipFree(temp);
}